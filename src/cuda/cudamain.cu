#include "hip/hip_runtime.h"
__device__ Color raytrace(const Ray& ray)
{
    // we use double for vectors, rays and so on and floats for colors
    const Node* closestNode = nullptr;
    double closestDist = INF;
    IntersectionInfo closestInfo;

    for (const auto& node : nodes)
    {
        IntersectionInfo info;
        if (!node._geometry->intersect(ray, info))
        {
            continue;
        }

        if (info._distance < closestDist)
        {
            closestDist = info._distance;
            closestNode = &node;
            closestInfo = info;
        }
    }

    // check if we hit the sky
    if (closestNode == nullptr)
    {
        return Color(0.f, 0.f, 0.f); // background color
    }
    else
    {
        return closestNode->_shader->shade(ray, closestInfo);
    }
}

__global__ void render(int width, int height)
{
    for (int y = 0; y < height; y++)
        for (int x = 0; x < width; x++) {
            Ray ray = camera.getScreenRay(x, y);
            vfb[y][x] = raytrace(ray);
        }
}